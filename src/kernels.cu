
#include <hip/hip_runtime.h>
#define Inf 9999999

void __global__ dark_channel_ker(float * image, float * darkc, float * index, int N, int H, int W, int wsize)
{
    int C = 3;
    int D = 1;
    int hsize = (wsize - 1) / 2;

    int ids = blockDim.x * blockIdx.x + threadIdx.x;
    int idt = gridDim.x * blockDim.x;

    for(; ids<W*H*N; ids+=idt)
    {
        // ids = W*H*n + W*h + w
        int idw = ids % W;
        ids = (ids - idw) / W;
        int idh = ids % H;
        ids = (ids - idh) / H;
        int idn = ids;

        float tmpc = Inf;
        float tmp1 = 0;
        float tmp2 = 0;
        float tmp3 = 0;

        for(int idc=0; idc<C; idc++)
        {
            for(int p=idh-hsize; p<idh+hsize+1; p++)
            {
                for(int q=idw-hsize; q<idw+hsize+1; q++)
                {
                    if(p>-1 && p<H && q>-1 && q<W)
                    {
                        float pixel_value = image[W*H*C*idn + W*H*idc + W*p + q];
                        if(pixel_value<tmpc)
                        {
                            tmpc = pixel_value;
                            tmp1 = idc;
                            tmp2 = p;
                            tmp3 = q;
                        }
                    }
                }
            }
        }

        darkc[W*H*D*idn + W*H*0 + W*idh + idw] = tmpc;
        index[W*H*C*idn + W*H*0 + W*idh + idw] = tmp1;
        index[W*H*C*idn + W*H*1 + W*idh + idw] = tmp2;
        index[W*H*C*idn + W*H*2 + W*idh + idw] = tmp3;
    }
}

void __global__ dark_extract_ker(float * image, float * index, float * darkc, int N, int H, int W)
{
    int C = 3;

    int ids = blockDim.x * blockIdx.x + threadIdx.x;
    int idt = gridDim.x * blockDim.x;

    for(; ids<N*H*W; ids+=idt)
    {
        int idw = ids % W;
        ids = (ids - idw) / W;
        int idh = ids % H;
        ids = (ids - idh) / H;
        int idn = ids;

        int idz = index[W*H*C*idn + W*H*0 + W*idh + idw];
        int idx = index[W*H*C*idn + W*H*1 + W*idh + idw];
        int idy = index[W*H*C*idn + W*H*2 + W*idh + idw];

        darkc[W*H*idn + W*idh + idw] = image[W*H*C*idn + W*H*idz + W*idx + idy];
    }
}

void __global__ place_back_ker(float * darkc, float * index, float * image, float * accum, int N, int H, int W, int wsize)
{
    int C = 3;
    int hsize = (wsize - 1) / 2;

    int ids = blockDim.x * blockIdx.x + threadIdx.x;
    int idt = gridDim.x * blockDim.x;

    for(; ids<N*C*H*W; ids+=idt)
    {
        int idw = ids % W;
        ids = (ids - idw) / W;
        int idh = ids % H;
        ids = (ids - idh) / H;
        int idc = ids % C;
        ids = (ids - idc) / C;
        int idn = ids;

        image[W*H*C*idn + W*H*idc + W*idh + idw] = 0;
        accum[W*H*C*idn + W*H*idc + W*idh + idw] = 0;
        float tmpi = 0;
        float tmpa = 0;

        for(int p=idh-hsize; p<idh+hsize+1; p++)
        {
            for(int q=idw-hsize; q<idw+hsize+1; q++)
            {
                if(p>-1 && p<H && q>-1 && q<W)
                {
                    int idz = (int) index[W*H*C*idn + W*H*0 + W*p + q];
                    int idx = (int) index[W*H*C*idn + W*H*1 + W*p + q];
                    int idy = (int) index[W*H*C*idn + W*H*2 + W*p + q];
                    if(idx==idh && idy==idw && idz==idc)
                    {
                        tmpi += darkc[W*H*idn + W*p + q];
                        tmpa += 1;
                    }
                }
            }
        }
        image[W*H*C*idn + W*H*idc + W*idh + idw] = tmpi;
        accum[W*H*C*idn + W*H*idc + W*idh + idw] = tmpa;
    }
}

#ifdef __cplusplus
extern "C"
{
    #endif

    void dark_channel_run(float * d_image, float * d_darkc, float * d_index, int N, int H, int W, int wsize)
    {
        int const threadsPerBlock = 1024;
        int const blocksPerGrid = (N*H*W + threadsPerBlock - 1) / threadsPerBlock;
        dark_channel_ker<<<blocksPerGrid, threadsPerBlock>>>(d_image, d_darkc, d_index, N, H, W, wsize);
    }

    void dark_extract_run(float * d_image, float * d_index, float * d_darkc, int N, int H, int W)
    {
        int const threadsPerBlock = 1024;
        int const blocksPerGrid = (N*H*W + threadsPerBlock - 1) / threadsPerBlock;
        dark_extract_ker<<<blocksPerGrid, threadsPerBlock>>>(d_image, d_index, d_darkc, N, H, W);
    }

    void place_back_run(float * d_darkc, float * d_index, float * d_image, float * d_accum, int N, int H, int W, int wsize)
    {
        int const threadsPerBlock = 1024;
        int const blocksPerGrid = (N*3*H*W + threadsPerBlock - 1) / threadsPerBlock;
        place_back_ker<<<blocksPerGrid, threadsPerBlock>>>(d_darkc, d_index, d_image, d_accum, N, H, W, wsize);
    }

    #ifdef __cplusplus
}
#endif
